/** @file     main.c
 *  @brief    Traveling Salesman Problem.
 *  @author   Marcelo Pinto (xmrcl0@gmail.com)
 *  @date     09/26/2017
 *  @version  0.3
 */

#include "tsp.h"
#include "print.h"
#include "utils.h"
#include "graphviz.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

void
help (void)
{
  printf ("usage: tsp [-h] [-g] [-n <ITER>] -m <MODE> -b <BNUM> -t <TNUM> -f <FILE>\n");
  printf ("Find best path to Traveling Salesman Problem using Monte Carlo Method\n\n");
  printf ("Options:\n");
  printf ("  -n <ITER>    Number of paths to simulate per thread\n");
  printf ("  -m <MODE>    Exibition mode 0, 1 or 2 (silent = 0)\n");
  printf ("  -b <BNUM>    Number of blocks in the grid\n");
  printf ("  -t <TNUM>    Number of threads per block(must be power of two)\n");
  printf ("  -f <FILE>    Cities coordinates file\n");
  printf ("  -g           Generate city coordinates + shortest path graph in graphviz's dot format\n");
  printf ("  -h           Show this help message and exit\n\n");
  printf ("Example:\n");
  printf ("  tspcuda -b 64 -t 256 -n 2000 -m 0 -f data/grid15_xy.txt   # Using 64 blocks of 256 threads, simulate 2000 paths in each thread for 15 cities data file\n");
}


int
parse_cmdline(int argc, char **argv, long double *num_iter, int *num_cities, float ***coord, int *mode, int *gendot, int *threadsPerBlock, int *numBlocks)
{
  char c;
  long double i;
  int nflag = 0, mflag = 0, fflag = 0, gflag = 0, bflag = 0, tflag = 0;
  float len = 0, min_len = FLT_MAX;
  FILE *file;


  // Read and parse command line arguments
  opterr = 0;
  while ((c = getopt (argc, argv, "n:m:f:b:t:gh::")) != -1)
    switch (c)
    {
    case 'n':
      nflag = 1;
      if (!is_integer (optarg))
      {
        fprintf (stderr, "%s: error: number of simulations must be an integer\n", argv[0]);
        exit (EXIT_FAILURE);
      }
      else
      {
        *num_iter = strtold (optarg, NULL);
      }
      break;
    case 'm':
      mflag = 1;
      if (!is_positive_number (optarg))
      {
        fprintf (stderr, "%s: error: invalid mode, choose 0, 1 or 2\n", argv[0]);
        exit (EXIT_FAILURE);
      }
      else
      {
        *mode = atoi (optarg);
      }
      if (*mode > 2)
      {
        fprintf (stderr, "%s: error: invalid mode, choose 0, 1 or 2\n", argv[0]);
        exit (EXIT_FAILURE);
      }
      break;
    case 'f':
      fflag = 1;
      *num_cities = read_file (optarg, coord);
      if (num_cities == 0)
      {
        fprintf (stderr, "%s: error: no such file or directory\n", argv[0]);
        exit (EXIT_FAILURE);
      }
      else
      if (*num_cities == -1)
      {
        fprintf (stderr, "%s: error: incompatible data file\n", argv[0]);
        exit (EXIT_FAILURE);
      }
      break;
    case 'g':
      gflag = 1;
      *gendot = 1;
      break;
    case 't':
      tflag = 1;
      if (!is_integer (optarg))
      {
        fprintf (stderr, "%s: error: number of threads per block must be an integer and power of two\n", argv[0]);
        exit (EXIT_FAILURE);
      }
      else
      {
        *threadsPerBlock = strtold (optarg, NULL);
      }
      break;
    case 'b':
      bflag = 1;
      if (!is_integer (optarg))
      {
        fprintf (stderr, "%s: error: number of blocks in the grid must be an integer\n", argv[0]);
        exit (EXIT_FAILURE);
      }
      else
      {
	*numBlocks = strtold (optarg, NULL);
      }
      break;
    case 'h':
      help ();
      exit (EXIT_SUCCESS);
      break;
    case '?':
      fprintf (stderr, "%s: error: invalid option\n", argv[0]);
      return 1;
    default:
      fprintf (stderr, "usage: tsp [-h] [-g] [-n <ITER>] -b <BNUM> -t <TNUM> -m <MODE> -f <FILE>\n");
      abort ();
    }

  for (i = optind; i < argc; i++)
  {
    fprintf (stderr, "%s: error: too many or too few arguments\n", argv[0]);
    exit (EXIT_FAILURE);
  }

  if (num_iter + 1 < num_iter)
  {
    fprintf (stderr, "%s: error: number of simulations must be less than %Lf \n", argv[0], LDBL_MAX);
    exit (EXIT_FAILURE);
  }

  // Check if obrigatory argumets were given
  if (nflag == 0 || mflag == 0 || fflag == 0)
  {
    fprintf (stderr, "%s: error: too few parameters\n", argv[0]);
    fprintf (stderr, "usage: tsp [-h] [-n <ITER>] -b <BNUM> -t <TNUM> -m <MODE> -f <FILE>\n");
    exit (EXIT_FAILURE);
  }
}

void
setupGPU (unsigned int device, struct hipDeviceProp_t *deviceProp, unsigned int *blocksize, unsigned int *gridsize)
{
  hipError_t cudaResult = hipSuccess;
  cudaResult = hipGetDeviceProperties(deviceProp, device);

  if (cudaResult != hipSuccess)
  {
    fprintf (stderr, "Não foi possível obter as propriedades da GPU.\n");
    fprintf (stderr, hipGetErrorString(cudaResult));
    exit (EXIT_FAILURE);
  }

  cudaResult = hipSetDevice(device);

  if (cudaResult != hipSuccess)
  {
    fprintf (stderr, "Não foi possível conectar à GPU.\n");
    fprintf (stderr, hipGetErrorString(cudaResult));
    exit (EXIT_FAILURE);
  }

  if (*blocksize > (unsigned int)deviceProp->maxThreadsDim[0])
  {
    fprintf (stderr, "O número de threads por bloco excede as capacidades do dispositivo");
    exit (EXIT_FAILURE);
  }

  if (*gridsize > (unsigned int)deviceProp->maxGridSize[0])
  {
    fprintf (stderr, "O número de blocos na grid excede as capacidades do dispositivo");
    exit (EXIT_FAILURE);
  }
}

int
main (int argc, char **argv)
{
  long double i, num_iter;
  int num_cities, mode, gendot = 0, threadsPerBlock = 256, numBlocks = 32;
  float **coord, **distance;
  int *min_path;
  float len = 0, min_len = FLT_MAX;

  // Parse command line
  parse_cmdline(argc, argv, &num_iter, &num_cities, &coord, &mode, &gendot, &threadsPerBlock, &numBlocks);
 

  // Create distance matrix
  distance_matrix (&coord, &distance, num_cities);

  //Hardcoded device for now
  unsigned int device = 0;
  hipError_t cudaResult = hipSuccess;
  
  //Block and grid
  dim3 block;
  dim3 grid;
  //Hardcoded for now
  block.x = threadsPerBlock;
  grid.x = numBlocks;
  
  //Initalize device, perform basic checks
  struct hipDeviceProp_t deviceProp;
  setupGPU (device, &deviceProp, &block.x, &grid.x);

  // Allocate memory for RNG states
  hiprandState *d_rngStates = 0;
  cudaResult = hipMalloc((void **)&d_rngStates, grid.x * block.x * sizeof(hiprandState));

  if (cudaResult != hipSuccess)
  {
    fprintf(stderr, "Erro: não foi possível alocar memóra na GPU para os estados do RNG\n");
    fprintf(stderr, hipGetErrorString(cudaResult));
    exit(EXIT_FAILURE);
  }
  
  //Initialize RNG
  initRNG<<<grid, block>>>(d_rngStates, time (NULL));

  //Sadly CUDA doesn't like arrays-of-pointers matrices very much, flattened coord and
  //distance matrices are thus needed.
  float *fdistance;
  distance_vector (&coord, &fdistance, num_cities);
  
  //Allocate and copy distance matrix to device
  float * d_distance;

  cudaResult = hipMalloc( (void **) &d_distance, num_cities * num_cities * sizeof(float));

  if (cudaResult != hipSuccess)
  {
    fprintf(stderr, "Erro: não foi possível alocar memóra na GPU para a matriz de distâncias\n");
    fprintf(stderr, hipGetErrorString(cudaResult));
    exit(EXIT_FAILURE);
  }

  cudaResult = hipMemcpy(d_distance, fdistance, num_cities * num_cities * sizeof(float), hipMemcpyHostToDevice);

  if (cudaResult != hipSuccess)
  {
    fprintf(stderr, "Erro: não foi possível copiar a matriz de distâncias para a GPU.\n");
    fprintf(stderr, hipGetErrorString(cudaResult));
    exit(EXIT_FAILURE);
  }

  //Free the flattened distance matrix
  free(fdistance);

  //Allocate memory in device for computation results
  int * d_minpaths;

  cudaResult = hipMalloc( (void **) &d_minpaths, grid.x * num_cities * sizeof(int));

  if (cudaResult != hipSuccess)
  {
    fprintf(stderr, "Erro: não foi possível alocar memória na GPU para os resultados(caminhos)\n");
    fprintf(stderr, hipGetErrorString(cudaResult));
    exit(EXIT_FAILURE);
  }

  float * d_mindists;

  cudaResult = hipMalloc( (void **) &d_mindists, grid.x * sizeof(float));

  if (cudaResult != hipSuccess)
  {
    fprintf(stderr, "Erro: não foi possível alocar memória na GPU para os resultados(distâncias)\n");
    fprintf(stderr, hipGetErrorString(cudaResult));
    exit(EXIT_FAILURE);
  }
  
  /* Shared memory setup:
   * - One float for each thread in a block to store the minimum distance computed
   * - Two num_cities-long int array. Threads alternate between using one for storing
   *   the best path and the other for storing the next path to be computed. That way
   *   one is able to avoid the horridly expensive memory copying I was doing earlier
   *   and possibly obtaing better memory locality in warps.
   */
  kernel<<<grid, block,
    block.x * sizeof(float) + 2 * block.x * sizeof(int) * num_cities>>>
    (d_mindists, d_minpaths, d_distance, d_rngStates,  num_cities, num_iter);

  // Copy results back to device
  float *mindists = (float *) malloc(grid.x * sizeof(float));
  cudaResult = hipMemcpy(mindists, d_mindists, grid.x * sizeof(float), hipMemcpyDeviceToHost);
  
  if (cudaResult != hipSuccess)
  {
    fprintf(stderr, "Erro: não foi possível copiar resultados(distâncias) para o host\n");
    fprintf(stderr, hipGetErrorString(cudaResult));
    exit(EXIT_FAILURE);
  }
  
  // Finish reduction on host
  int min_idx = 0;
  min_len = mindists[0];
  for (int i = 1; i < grid.x; i++)
  {
    if (mindists[i] < min_len)
    {
      min_len = mindists[i];
      min_idx = i;
    }
  }
  free(mindists);

  min_path = (int *) malloc(num_cities * sizeof(int));

  cudaResult = hipMemcpy(min_path, &d_minpaths[min_idx * num_cities], num_cities * sizeof(int), hipMemcpyDeviceToHost);
  
  if (cudaResult != hipSuccess)
  {
    fprintf(stderr, "Erro: não foi possível copiar melhor caminho para o host\n");
    fprintf(stderr, hipGetErrorString(cudaResult));
    exit(EXIT_FAILURE);
  }

  // Clean up device variables
  hipFree(d_rngStates);
  hipFree(d_distance);
  hipFree(d_minpaths);
  hipFree(d_mindists);


  // Print report 
  print_repo (coord, distance, min_path, num_cities, min_len, num_iter, mode);

  // Generate dot file
  if(gendot) {
      gen_graphviz (coord, min_path, num_cities);
  }
  
  free (min_path);
  free (coord);
  free (distance);

  return 0;
}
