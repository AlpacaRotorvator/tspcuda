/** @file     utils.c
 *  @brief    Utils functions.
 *  @author   Marcelo Pinto (xmrcl0@gmail.com)
 *  @date     09/12/2017
 */

#include <hip/hip_runtime.h>
#include "utils.h"

void
handleCudaErrors (hipError_t cudaResult)
{
  if (cudaResult != hipSuccess) {
    fprintf (stderr, hipGetErrorString(cudaResult));
    exit (EXIT_FAILURE);
  }
}

int
is_coordinate (char *line)
{
  int r;
  regex_t regex;
  const char *pattern = "^[0-9]+\\.?([0-9]+)?[ \t]+[0-9]+\\.?([0-9]+)?[ \t]+?\n";

  r = regcomp (&regex, pattern, REG_EXTENDED);
  if (r)
  {
    fprintf (stderr, "Could not compile regex\n");
    return -1;
  }

  r = regexec (&regex, line, 0, NULL, 0);
  if (!r)
  {
    return 1;
  }
  return 0;
}


int
is_integer (char *number)
{
  int r;
  regex_t regex;
  const char *pattern = "^[0-9]+$";

  r = regcomp (&regex, pattern, REG_EXTENDED);
  if (r)
  {
    fprintf (stderr, "Could not compile regex\n");
    return -1;
  }

  r = regexec (&regex, number, 0, NULL, 0);
  if (!r)
  {
    return 1;
  }
  return 0;
}


int
is_positive_number (char *number)
{
  int r;
  regex_t regex;
  const char *pattern = "^[0-9]+\\.?([0-9]+)?$";

  r = regcomp (&regex, pattern, REG_EXTENDED);
  if (r)
  {
    fprintf (stderr, "Could not compile regex\n");
    return -1;
  }

  r = regexec (&regex, number, 0, NULL, 0);
  if (!r)
  {
    return 1;
  }
  return 0;
}


void
array_copy (int **src, int **dst, size_t n)
{
  int i;

  (*dst) = (int *) malloc ((n + 1) * sizeof (int));

  for (i = 0; i <= n; i++)
    (*dst)[i] = (*src)[i];
}


long double
factorial (int n)
{
  int i;
  long double factorial = 1;

  for (i = n; i > 1; i--)
    factorial = factorial * i;
  return factorial;
}
