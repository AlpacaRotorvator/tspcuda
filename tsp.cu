#include "hip/hip_runtime.h"
/** @file     tsp.c
 *  @brief    Traveling Salesman Problem functions.
 *  @author   Marcelo Pinto (xmrcl0@gmail.com)
 *  @date     09/12/2017
 *  @updated  09/21/2017
 */

#include "tsp.h"
#include "print.h"
#include "utils.h"


void
distance_matrix (float ***coord, float ***distance, int num_city)
{
  int i, j, nrows, ncols;

  ncols = num_city;
  nrows = num_city;

  *distance = (float **) malloc (nrows * sizeof (float *));
  for (i = 0; i < nrows; i++)
    (*distance)[i] = (float *) malloc (ncols * sizeof (float));

  for (i = 0; i < num_city; i++)
    for (j = 0; j < num_city; j++)
      (*distance)[i][j] = sqrt (pow ((*coord)[i][0] - (*coord)[j][0], 2) + pow ((*coord)[i][1] - (*coord)[j][1], 2));
}

void
distance_vector (float ***coord, float **distance, int num_city)
{
  int i, j, nrows, ncols;

  ncols = num_city;
  nrows = num_city;

  *distance = (float *) malloc (num_city * num_city  * sizeof (float));

  for (i = 0; i < num_city; i++)
  {
    for (j = 0; j < num_city; j++)
    {
      (*distance)[i + j * num_city] = sqrt (pow ((*coord)[i][0] - (*coord)[j][0], 2) + pow ((*coord)[i][1] - (*coord)[j][1], 2));
    }
  }
}

__global__ void
initRNG(hiprandState *const rngStates, const unsigned int seed)
{
    // Determine thread ID
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Initialise the RNG
    hiprand_init(seed, tid, 0, &rngStates[tid]);
}

void
create_path (int num_city, int **coord)
{
  int i;

  (*coord) = (int *) malloc ((num_city + 1) * sizeof (int));

  randperm (num_city, *coord);
  (*coord)[num_city] = (*coord)[0];
}


float
measure_path (float ***distance, int num_city, int **path)
{
  int i;
  float l = 0;

  for (i = 0; i < num_city; i++)
    l = l + (*distance)[(*path)[i]][(*path)[i + 1]];
  return l;
}


int
read_file (char *file, float ***array)
{
  int i, j, nrows = 0, ncols = 2;
  char c;
  char *line = NULL;
  size_t len=0;
  FILE *fp;

  fp = fopen (file, "r");
  if (fp == NULL)
    return 0;

  while ((getline(&line, &len, fp) != -1))
  { 
    if (!is_coordinate (line))
      return -1;
    nrows++;
  }
  free(line);

  // Allocate memory for coordinates matrix 
  *array = (float **) malloc (nrows * sizeof (float *));
  for (i = 0; i < nrows; i++)
    (*array)[i] = (float *) malloc (ncols * sizeof (float));

  // Read coordinates from file to coordinates matrix
  fseek (fp, 0, SEEK_SET);
  for (i = 0; i < nrows; i++)
    for (j = 0; j < ncols; j++)
      if (!fscanf (fp, "%f", &(*array)[i][j]))
	break;
  fclose (fp);

  return nrows;
}
