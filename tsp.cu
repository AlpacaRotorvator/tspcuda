#include "hip/hip_runtime.h"
/** @file     tsp.c
 *  @brief    Traveling Salesman Problem functions.
 *  @author   Marcelo Pinto (xmrcl0@gmail.com)
 *  @date     09/12/2017
 *  @updated  09/21/2017
 */

#include "tsp.h"
#include "print.h"
#include "utils.h"


void
distance_matrix (float ***coord, float ***distance, int num_city)
{
  int i, j, nrows, ncols;

  ncols = num_city;
  nrows = num_city;

  *distance = (float **) malloc (nrows * sizeof (float *));
  for (i = 0; i < nrows; i++)
    (*distance)[i] = (float *) malloc (ncols * sizeof (float));

  for (i = 0; i < num_city; i++)
    for (j = 0; j < num_city; j++)
      (*distance)[i][j] = sqrt (pow ((*coord)[i][0] - (*coord)[j][0], 2) + pow ((*coord)[i][1] - (*coord)[j][1], 2));
}

void
distance_vector (float ***coord, float **distance, int num_city)
{
  int i, j, nrows, ncols;

  ncols = num_city;
  nrows = num_city;

  *distance = (float *) malloc (num_city * num_city  * sizeof (float));

  for (i = 0; i < num_city; i++)
  {
    for (j = 0; j < num_city; j++)
    {
      (*distance)[i + j * num_city] = sqrt (pow ((*coord)[i][0] - (*coord)[j][0], 2) + pow ((*coord)[i][1] - (*coord)[j][1], 2));
    }
  }
}

__global__ void
initRNG(hiprandState *const rngStates, const unsigned int seed)
{
    // Determine thread ID
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Initialise the RNG
    hiprand_init(seed, tid, 0, &rngStates[tid]);
}

__device__ void
create_path (int num_city, int *coord, hiprandState localState)
{
  randperm (num_city, coord, localState);
}


__device__ float
measure_path (float *distance, int num_city, int *path)
{
  int i;
  float l = 0;

  for (i = 0; i < num_city - 1; i++)
  {
    int j = path[i];
    int k = path[i + 1];
    l = l + distance[j + num_city * k];
  }
  return l;
}

__global__ void
kernel (float *const mindists, int *const minpaths, float *const distance,
	hiprandState *const rngStates, const int n_cities, const int n_iter)
{
  // Determine thread ID
  unsigned int bid = blockIdx.x;
  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int ltid = threadIdx.x;

  // Sort out shared memory
  extern __shared__ float sdata[];
  float *threadsMinDists = sdata;
  int *minPathMatrix = (int *) &threadsMinDists[blockDim.x];
  int *computePathMatrix = (int *) &minPathMatrix[n_cities * blockDim.x];

  // Sort out local(ie this thread's) variables
  float *curThreadMinDist = &threadsMinDists[ltid];
  int *curThreadMinPath = &minPathMatrix[ltid * n_cities];
  int *curThreadCptPath = &computePathMatrix[ltid * n_cities];
  hiprandState localState = rngStates[tid];

  //Run everything at least once to initialize a sane minimum path
  create_path (n_cities, curThreadMinPath, localState);
  *curThreadMinDist =  measure_path (distance, n_cities, curThreadMinPath);

  float curThreadCptDist = 0;
  for (int i = 1; i < n_iter; i++)
  {
    create_path (n_cities, curThreadCptPath, localState);
    curThreadCptDist = measure_path (distance, n_cities, curThreadMinPath);

    if (curThreadCptDist < *curThreadMinDist)
    {
      *curThreadMinDist = curThreadCptDist;
      // This is not great at all for performance but guess who cares?
      memcpy (curThreadMinPath, curThreadCptPath, sizeof(int) * n_cities);
    }
  }
  unsigned int minDistTid = reduce_dists(threadsMinDists);
  if(ltid == 0)
  {
    printf("\n\t%d\n", minDistTid);
  }
  
  if (ltid == minDistTid)
  {
    mindists[bid] = threadsMinDists[0];
    if(bid == 2)
    {
      printf("\n(%d,%d) here! My distance is %f, and my path:\n\t[", bid, ltid, mindists[bid]);
      for(int i = 0; i < n_cities - 1; i++)
      {
	printf("%d, ", curThreadMinPath[i]);
      }
      printf("%d]\n", curThreadMinPath[n_cities - 1]);
    }
    memcpy(curThreadMinPath, &minpaths[bid], sizeof(int) * n_cities);
  }
}

__device__ unsigned int
reduce_dists(float *const threadsMinDists)
{
  unsigned int ltid = threadIdx.x;
  
  __syncthreads();
  
  // Do reduction in shared mem
  for (unsigned int s = blockDim.x / 2 ; s > 0 ; s >>= 1)
  {
    if (ltid < s)
    {
      if (threadsMinDists[s] < threadsMinDists[ltid])
      {
	threadsMinDists[ltid] = threadsMinDists[ltid + s];
	threadsMinDists[ltid + s] = ltid + s;
      }
      else {
	threadsMinDists[ltid + s] = ltid;
      }
    }
    __syncthreads();
  }

    return threadsMinDists[1];
}

int
read_file (char *file, float ***array)
{
  int i, j, nrows = 0, ncols = 2;
  char c;
  char *line = NULL;
  size_t len=0;
  FILE *fp;

  fp = fopen (file, "r");
  if (fp == NULL)
    return 0;

  while ((getline(&line, &len, fp) != -1))
  { 
    if (!is_coordinate (line))
      return -1;
    nrows++;
  }
  free(line);

  // Allocate memory for coordinates matrix 
  *array = (float **) malloc (nrows * sizeof (float *));
  for (i = 0; i < nrows; i++)
    (*array)[i] = (float *) malloc (ncols * sizeof (float));

  // Read coordinates from file to coordinates matrix
  fseek (fp, 0, SEEK_SET);
  for (i = 0; i < nrows; i++)
    for (j = 0; j < ncols; j++)
      if (!fscanf (fp, "%f", &(*array)[i][j]))
	break;
  fclose (fp);

  return nrows;
}

__device__ void
randperm (int n, int perm[], hiprandState localState)
{
  int i, j, t;

  for (i = 0; i < n; i++)
  {
    perm[i] = i;
  }
  for (i = 0; i < n; i++)
  {
    j = hiprand (&localState) % (n - i) + i;
    t = perm[j];
    perm[j] = perm[i];
    perm[i] = t;
  }
}
